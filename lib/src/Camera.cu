#include "hip/hip_runtime.h"
#include "Camera.cuh"


__host__ Camera::Camera(float2 fov, float3 camPos, float3 camRot)
    : fov(fov), camPos(camPos), camRot(camRot)
    {}