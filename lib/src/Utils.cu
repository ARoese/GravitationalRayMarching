//
// Created by atomr on 7/14/2025.
//

#include "Utils.cuh"
void wrap_cuda(std::function<hipError_t()> action) {
    auto res = action();
    if (res != hipSuccess) {
        throw res;
    }
}
