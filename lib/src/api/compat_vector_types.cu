#include "hip/hip_runtime.h"
//
// Created by atomr on 7/19/2025.
//
#include "compat_vector_types.cuh"
uchar3 fromCompat(const d_uchar3& o) {
    return uchar3(o.x, o.y, o.z);
}

uint2 fromCompat( const d_uint2& o) {
    return uint2(o.x, o.y);
}

float3 fromCompat(const d_float3& o) {
    return float3(o.x, o.y, o.z);
}

float2 fromCompat(const d_float2& o) {
    return float2(o.x, o.y);
}